
#include <hip/hip_runtime.h>
#include <stdio.h>

// V1: Single channel, no batching, fairly naive shared memory implementation

// error checking macro
#define cudaCheckErrors(msg)                                   \
    do                                                         \
    {                                                          \
        hipError_t __err = hipGetLastError();                \
        if (__err != hipSuccess)                              \
        {                                                      \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                    msg, hipGetErrorString(__err),            \
                    __FILE__, __LINE__);                       \
            fprintf(stderr, "*** FAILED - ABORTING\n");        \
            exit(1);                                           \
        }                                                      \
    } while (0)

const int DSIZE = 1 << 10;
const int KERNEL_SIZE = 3;
const int HALO = KERNEL_SIZE / 2;
const int BLOCK_SIZE = 16;

__global__ void conv2d(int *in, int *out, int *kernel)
{
    int gidx = threadIdx.x + blockIdx.x * blockDim.x + HALO;
    int gidy = threadIdx.y + blockIdx.y * blockDim.y + HALO;
    if (gidy + HALO > DSIZE || gidx + HALO > DSIZE) return;

    int lidx = threadIdx.x + HALO;
    int lidy = threadIdx.y + HALO;

    // Make local copy of input tile (larger than output tile)
    __shared__ int lkernel[KERNEL_SIZE][KERNEL_SIZE];
    __shared__ int lin[BLOCK_SIZE + 2 * HALO][BLOCK_SIZE + 2 * HALO];

    // Fill centre of tile
    lin[lidx][lidy] = in[gidx + DSIZE * gidy];

    // Fill left edge of tile
    if (threadIdx.x < HALO)
    {
        lin[lidx - HALO][lidy] = in[gidx - HALO + DSIZE * gidy];

        // Fill top left corner
        if (threadIdx.y < HALO)
        {
            lin[lidx - HALO][lidy - HALO] = in[gidx - HALO + DSIZE * (gidy - HALO)];
        }

        // Fill bottom left corner
        if (threadIdx.y >= blockDim.y - HALO)
        {
            lin[lidx - HALO][lidy + HALO] = in[gidx - HALO + DSIZE * (gidy + HALO)];
        }
    }

    // Fill right edge of tile
    if (threadIdx.x >= blockDim.x - HALO)
    {
        lin[lidx + HALO][lidy] = in[gidx + HALO + DSIZE * gidy];

        // Fill top right corner
        if (threadIdx.y < HALO)
        {
            lin[lidx + HALO][lidy - HALO] = in[gidx + HALO + DSIZE * (gidy - HALO)];
        }

        // Fill bottom right corner
        if (threadIdx.y >= blockDim.y - HALO)
        {
            lin[lidx + HALO][lidy + HALO] = in[gidx + HALO + DSIZE * (gidy + HALO)];
        }
    }

    // Fill top of tile
    if (threadIdx.y < HALO)
    {
        lin[lidx][lidy - HALO] = in[gidx + DSIZE * (gidy - HALO)];
    }

    // Fill bottom of tile
    if (threadIdx.y >= blockDim.y - HALO)
    {
        lin[lidx][lidy + HALO] = in[gidx + DSIZE * (gidy + HALO)];
    }

    // Make local copy of kernel
    if (threadIdx.x < KERNEL_SIZE && threadIdx.y < KERNEL_SIZE)
    {
        lkernel[threadIdx.x][threadIdx.y] = kernel[threadIdx.x + KERNEL_SIZE * threadIdx.y];
    }

    __syncthreads();

    // Do the calculation
    int value = 0;
    for (int i = 0; i < KERNEL_SIZE; ++i)
    {
        for (int j = 0; j < KERNEL_SIZE; ++j)
        {
            value += lkernel[i][j] * lin[lidx + i - HALO][lidy + j - HALO];
        }
    }

    // Store the result
    out[gidx + DSIZE * gidy] = value;
}

int main()
{
    int *h_in = new int[DSIZE * DSIZE];
    int *h_out = new int[DSIZE * DSIZE];
    int *h_kernel = new int[KERNEL_SIZE * KERNEL_SIZE];

    for (int i = 0; i < DSIZE * DSIZE; ++i)
    {
        h_in[i] = 1;
        h_out[i] = 0;
    }
    for (int i = 0; i < KERNEL_SIZE * KERNEL_SIZE; ++i)
    {
        h_kernel[i] = 1;
    }

    int *d_in;
    int *d_out;
    int *d_kernel;

    hipMalloc(&d_in, DSIZE * DSIZE * sizeof(int));
    hipMalloc(&d_out, DSIZE * DSIZE * sizeof(int));
    hipMalloc(&d_kernel, KERNEL_SIZE * KERNEL_SIZE * sizeof(int));
    cudaCheckErrors("hipMalloc failure");

    hipMemcpy(d_in, h_in, DSIZE * DSIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out, h_out, DSIZE * DSIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, h_kernel, KERNEL_SIZE * KERNEL_SIZE * sizeof(int), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");

    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((DSIZE + block.x - 1) / block.x, (DSIZE + block.y - 1) / block.y);
    conv2d<<<grid, block>>>(d_in, d_out, d_kernel);
    cudaCheckErrors("Kernel launch failure");

    hipMemcpy(h_out, d_out, DSIZE * DSIZE * sizeof(int), hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy D2H failure");

    // Test result
    for (int i = HALO; i < DSIZE - HALO; ++i)
    {
        for (int j = HALO; j < DSIZE - HALO; ++j)
        {
            if (h_out[i + DSIZE * j] != KERNEL_SIZE * KERNEL_SIZE)
            {
                printf("mismatch at index %i, %i, was: %i, should be: %i\n",
                       i, j, h_out[i + DSIZE * j], KERNEL_SIZE * KERNEL_SIZE);
                return -1;
            }
        }
    }

    return 0;
}